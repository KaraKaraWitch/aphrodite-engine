#include "hip/hip_runtime.h"
#include <ATen/cuda/HIPContext.h>
#include <torch/extension.h>

#include "dispatch_utils.h"
#include "utils.cuh"
#include "reduction.cuh"

namespace aphrodite {
// TODO: Further optimize this kernel.
template<typename scalar_t>
__global__ void rms_norm_kernel(
  scalar_t* __restrict__ out,             // [..., hidden_size]
  const scalar_t* __restrict__ input,     // [..., hidden_size]
  const scalar_t* __restrict__ weight,    // [hidden_size]
  const float epsilon,
  const int num_tokens,
  const int hidden_size) {
  __shared__ float s_variance;
  float variance = 0.0f;

  for (int idx = threadIdx.x; idx < hidden_size; idx += blockDim.x) {
    const float x = (float)input[blockIdx.x * hidden_size + idx];
    variance += x * x;
  }
  variance = blockReduceSum<float>(variance);
  if (threadIdx.x == 0) {
    s_variance = rsqrtf(variance / hidden_size + epsilon);
  }
  __syncthreads();

  for (int idx = threadIdx.x; idx < hidden_size; idx += blockDim.x) {
    float x = (float)input[blockIdx.x * hidden_size + idx];
    out[blockIdx.x * hidden_size + idx] =
        ((scalar_t)(x * s_variance)) * weight[idx];
  }
}

template <typename T>
__global__ void rms_norm_quant_kernel(const T *__restrict__ input,
                                      const T *__restrict__ gamma,
                                      int8_t *__restrict__ output,
                                      const float layernorm_eps, int m, int n) {
  // layernorm module in the T5 style No bias and no subtraction of mean.
  const int tid = threadIdx.x;

  __shared__ float s_variance;
  float variance = 0.0f;

  float local_var_sum = 0.0f;
  for (int i = tid; i < n; i += blockDim.x) {
    // float diff = (float)(ldg(&input[blockIdx.x * n + i]));
    float diff = (float)(input[blockIdx.x * n + i]);
    local_var_sum += diff * diff;
  }
  variance = blockReduceSum(local_var_sum);

  if (threadIdx.x == 0) {
    s_variance = rsqrtf(variance / (float)n + layernorm_eps);
  }
  __syncthreads();

  for (int i = tid; i < n; i += blockDim.x) {
    output[blockIdx.x * n + i] = float_to_int8_rn(
        (((float)input[blockIdx.x * n + i]) * s_variance) * (float)(gamma[i]));
  }
}

template <typename T>
__global__ void dequant_add_residual_rms_norm_quant_kernel(
    const int32_t *__restrict__ input, T *__restrict__ residual,
    int8_t *__restrict__ output, const T *__restrict__ gamma,
    const float layernorm_eps, const float scale, int m, int n) {
  // layernorm module in the T5 style No bias and no subtraction of mean.
  const int tid = threadIdx.x;

  __shared__ float s_variance;
  float variance = 0.0f;

  float local_var_sum = 0.0f;
  for (int i = tid; i < n; i += blockDim.x) {
    float diff = ((((float)input[blockIdx.x * n + i]) * scale) +
                  (float)residual[blockIdx.x * n + i]);
    residual[blockIdx.x * n + i] = (T)diff;
    local_var_sum += diff * diff;
  }
  variance = blockReduceSum(local_var_sum);

  if (threadIdx.x == 0) {
    s_variance = rsqrtf(variance / (float)n + layernorm_eps);
  }
  __syncthreads();

  for (int i = tid; i < n; i += blockDim.x) {
    output[blockIdx.x * n + i] =
        float_to_int8_rn((((float)(residual[blockIdx.x * n + i])) * s_variance) * (float)(gamma[i]));
  }
}

} // namespace aphrodite


void rms_norm(
  torch::Tensor& out,      // [..., hidden_size]
  torch::Tensor& input,    // [..., hidden_size]
  torch::Tensor& weight,   // [hidden_size]
  float epsilon) {
  int hidden_size = input.size(-1);
  int num_tokens = input.numel() / hidden_size;

  dim3 grid(num_tokens);
  dim3 block(std::min(hidden_size, 1024));
  const hipStream_t stream = at::cuda::getCurrentCUDAStream();
  APHRODITE_DISPATCH_FLOATING_TYPES(input.scalar_type(), "rms_norm_kernel", [&] {
    aphrodite::rms_norm_kernel<scalar_t><<<grid, block, 0, stream>>>(
        out.data_ptr<scalar_t>(), input.data_ptr<scalar_t>(),
        weight.data_ptr<scalar_t>(), epsilon, num_tokens, hidden_size);
  });
}

void invoke_rms_norm_quant(torch::Tensor &out,   // [num_tokens, hidden_size]
                           torch::Tensor &input, // [num_tokens, hidden_size]
                           torch::Tensor &gamma, // [hidden_size]
                           float epsilon) {
  int m = input.size(0);
  int n = input.size(1);
  dim3 grid(m);
  dim3 block(min(n, 1024));

  const hipStream_t stream = at::cuda::getCurrentCUDAStream();
  APHRODITE_DISPATCH_FLOATING_TYPES(
      input.scalar_type(), "rms_norm_quant_kernel", [&] {
        aphrodite::rms_norm_quant_kernel<scalar_t><<<grid, block, 0, stream>>>(
            input.data_ptr<scalar_t>(), gamma.data_ptr<scalar_t>(),
            out.data_ptr<int8_t>(), epsilon, m, n);
      });
}

void invoke_dequant_add_residual_rms_norm_quant(
    torch::Tensor &out,      // [num_tokens, hidden_size]
    torch::Tensor &input,    // [num_tokens, hidden_size]
    torch::Tensor &residual, // [num_tokens, hidden_size]
    torch::Tensor &gamma,    // [hidden_size]
    float epsilon, float scale) {
  int m = input.size(0);
  int n = input.size(1);
  dim3 grid(m);
  dim3 block(min(n, 1024));

  const hipStream_t stream = at::cuda::getCurrentCUDAStream();
  APHRODITE_DISPATCH_FLOATING_TYPES(
      residual.scalar_type(), "dequant_add_residual_rms_norm_quant_kernel",
      [&] {
        aphrodite::dequant_add_residual_rms_norm_quant_kernel<scalar_t>
            <<<grid, block, 0, stream>>>(
                input.data_ptr<int32_t>(), residual.data_ptr<scalar_t>(),
                out.data_ptr<int8_t>(), gamma.data_ptr<scalar_t>(), epsilon,
                scale, m, n);
      });
}