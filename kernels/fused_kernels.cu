#include "hip/hip_runtime.h"
#include <ATen/cuda/HIPContext.h>
#include <torch/extension.h>

#include "dispatch_utils.h"
#include "utils.cuh"
#include <cassert>

namespace aphrodite {
template <typename T>
__global__ void dequant_add_residual_kernel(const int32_t *__restrict__ input,
                                            const T *__restrict__ residual,
                                            T *__restrict__ output,
                                            const float scale, int m, int n) {
  const int tid = threadIdx.x;
  for (int i = tid; i < n; i += blockDim.x) {
    output[blockIdx.x * n + i] =
        (T)((((float)input[blockIdx.x * n + i]) * scale) +
            (float)residual[blockIdx.x * n + i]);
  }
}

template <typename T>
__global__ void dequant_kernel(const int32_t *__restrict__ input,
                              T *__restrict__ output,
                              const float scale, int m, int n, int input_stride, int out_stride) {
  const int tid = threadIdx.x;
  for (int i = tid; i < n; i += blockDim.x) {
    output[blockIdx.x * out_stride + i] =
        (T)(((float)input[blockIdx.x * input_stride + i]) * scale);
  }
}

template <typename T>
__global__ void quant_kernel(const T *__restrict__ input,
                              int8_t *__restrict__ output,
                              const float scale, int m, int n) {
  const int tid = threadIdx.x;
  for (int i = tid; i < n; i += blockDim.x) {
    output[blockIdx.x * n + i] =
        float_to_int8_rn(((float)input[blockIdx.x * n + i]) / scale);
  }
}
} // namespace aphrodite

void invoke_dequant_add_residual(
    torch::Tensor &out,      // [num_tokens, hidden_size]
    torch::Tensor &input,    // [num_tokens, hidden_size]
    torch::Tensor &residual, // [num_tokens, hidden_size]
    float scale) {
  int m = input.size(0);
  int n = input.size(1);
  dim3 grid(m);
  dim3 block(min(n, 1024));

  const hipStream_t stream = at::cuda::getCurrentCUDAStream();
  APHRODITE_DISPATCH_FLOATING_TYPES(
      residual.scalar_type(), "dequant_add_residual_kernel", [&] {
        aphrodite::dequant_add_residual_kernel<scalar_t><<<grid, block, 0, stream>>>(
            input.data_ptr<int32_t>(), residual.data_ptr<scalar_t>(),
            out.data_ptr<scalar_t>(), scale, m, n);
      });
}

void invoke_dequant(
    torch::Tensor &out,      // [num_tokens, hidden_size]
    torch::Tensor &input,    // [num_tokens, hidden_size]
    float scale) {
  int m = input.size(0);
  int n = input.size(1);
  int input_stride = input.stride(0);
  int out_stride = out.stride(0);
  dim3 grid(m);
  dim3 block(min(n, 1024));

  const hipStream_t stream = at::cuda::getCurrentCUDAStream();
  APHRODITE_DISPATCH_FLOATING_TYPES(
      out.scalar_type(), "dequant_kernel", [&] {
        aphrodite::dequant_kernel<scalar_t><<<grid, block, 0, stream>>>(
            input.data_ptr<int32_t>(), out.data_ptr<scalar_t>(), scale, m, n, input_stride, out_stride);
      });
}

void invoke_quant(
    torch::Tensor &out,      // [num_tokens, hidden_size]
    torch::Tensor &input,    // [num_tokens, hidden_size]
    float scale) {
  assert(input.is_contiguous());
  assert(out.is_contiguous());
  int m = input.size(0);
  int n = input.size(1);
  dim3 grid(m);
  dim3 block(min(n, 1024));

  const hipStream_t stream = at::cuda::getCurrentCUDAStream();
  APHRODITE_DISPATCH_FLOATING_TYPES(
      input.scalar_type(), "quant_kernel", [&] {
        aphrodite::quant_kernel<scalar_t><<<grid, block, 0, stream>>>(
            input.data_ptr<scalar_t>(), out.data_ptr<int8_t>(), scale, m, n);
      });
}