#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <ATen/cuda/HIPContext.h>
#include "dispatch_utils.h"
namespace aphrodite {
template<typename scalar_t, bool IS_NEOX>
inline __device__ void apply_rotary_embedding(
  scalar_t* __restrict__ arr,
  const scalar_t* __restrict__ cos_ptr,
  const scalar_t* __restrict__ sin_ptr,
  int rot_offset,
  int embed_dim)
{
  int x_index, y_index;
  scalar_t cos, sin;
  if (IS_NEOX) {
    // GPT-NeoX style rotary embedding.
    x_index = rot_offset;
    y_index = embed_dim + rot_offset;
    cos = __ldg(cos_ptr + x_index);
    sin = __ldg(sin_ptr + x_index);
  } else {
    // GPT-J style rotary embedding.
    x_index = 2 * rot_offset;
    y_index = 2 * rot_offset + 1;
    cos = __ldg(cos_ptr + x_index / 2);
    sin = __ldg(sin_ptr + x_index / 2);
  }
  const scalar_t x = arr[x_index];
  const scalar_t y = arr[y_index];
  arr[x_index] = x * cos - y * sin;
  arr[y_index] = y * cos + x * sin;
}

template<typename scalar_t, bool IS_NEOX>
inline __device__ void apply_dequant_rotary_embedding(
  const int32_t* __restrict__ arr,
  scalar_t* __restrict__ arr_out,
  const scalar_t* __restrict__ cos_ptr,
  const scalar_t* __restrict__ sin_ptr,
  int rot_offset,
  int embed_dim,
  const float scale)
{
  int x_index, y_index;
  scalar_t cos, sin;
  if (IS_NEOX) {
    // GPT-NeoX style rotary embedding.
    x_index = rot_offset;
    y_index = embed_dim + rot_offset;
    cos = __ldg(cos_ptr + x_index);
    sin = __ldg(sin_ptr + x_index);
  } else {
    // GPT-J style rotary embedding.
    x_index = 2 * rot_offset;
    y_index = 2 * rot_offset + 1;
    cos = __ldg(cos_ptr + x_index / 2);
    sin = __ldg(sin_ptr + x_index / 2);
  }

  const scalar_t x = (scalar_t)((float)arr[x_index] * scale);
  const scalar_t y = (scalar_t)((float)arr[y_index] * scale);
  arr_out[x_index] = x * cos - y * sin;
  arr_out[y_index] = y * cos + x * sin;
}

template<typename scalar_t, bool IS_NEOX>
__global__ void rotary_embedding_kernel(
  const int64_t* __restrict__ positions,        // [batch_size, seq_len] or [num_tokens]
  scalar_t* __restrict__ query,                 // [batch_size, seq_len, num_heads, head_size] or [num_tokens, num_heads, head_size]
  scalar_t* __restrict__ key,                   // [batch_size, seq_len, num_kv_heads, head_size] or [num_tokens, num_kv_heads, head_size]
  const scalar_t* __restrict__ cos_sin_cache,   // [max_position, 2, rot_dim // 2]
  const int rot_dim,
  const int query_stride,
  const int key_stride,
  const int num_heads,
  const int num_kv_heads,
  const int head_size) {
  // Each thread block is responsible for one token.
  const int token_idx = blockIdx.x;
  int64_t pos = positions[token_idx];
  const scalar_t* cache_ptr = cos_sin_cache + pos * rot_dim;
  const int embed_dim = rot_dim / 2;
  const scalar_t* cos_ptr = cache_ptr;
  const scalar_t* sin_ptr = cache_ptr + embed_dim;
  const int nq = num_heads * embed_dim;
  for (int i = threadIdx.x; i < nq; i += blockDim.x) {
    const int head_idx = i / embed_dim;
    const int token_head = token_idx * query_stride + head_idx * head_size;
    const int rot_offset = i % embed_dim;
    apply_rotary_embedding<scalar_t, IS_NEOX>(query + token_head, cos_ptr,
                                              sin_ptr, rot_offset, embed_dim);
  }
  const int nk = num_kv_heads * embed_dim;
  for (int i = threadIdx.x; i < nk; i += blockDim.x) {
    const int head_idx = i / embed_dim;
    const int token_head = token_idx * key_stride + head_idx * head_size;
    const int rot_offset = i % embed_dim;
    apply_rotary_embedding<scalar_t, IS_NEOX>(key + token_head, cos_ptr,
                                              sin_ptr, rot_offset, embed_dim);
  }
}

template<typename scalar_t, bool IS_NEOX>
__global__ void dequant_rotary_embedding_kernel(
  const int64_t* __restrict__ positions,        // [num_tokens]
  const int32_t* __restrict__ query,                 // [num_tokens, num_heads, head_size]
  scalar_t* __restrict__ query_out,                 // [num_tokens, num_heads, head_size]
  const int32_t* __restrict__ key,                   // [num_tokens, num_kv_heads, head_size]
  scalar_t* __restrict__ key_out,                   // [num_tokens, num_kv_heads, head_size]
  const scalar_t* __restrict__ cos_sin_cache,   // [max_position, 2, rot_dim // 2]
  const int rot_dim,
  const int query_stride,
  const int query_out_stride,
  const int key_stride,
  const int key_out_stride,
  const int num_heads,
  const int num_kv_heads,
  const int head_size,
  const float query_scale,
  const float key_scale) {
  // Each thread block is responsible for one token.
  const int token_idx = blockIdx.x;
  int64_t pos = positions[token_idx];
  const scalar_t* cache_ptr = cos_sin_cache + pos * rot_dim;

  const int embed_dim = rot_dim / 2;
  const scalar_t* cos_ptr = cache_ptr;
  const scalar_t* sin_ptr = cache_ptr + embed_dim;

  const int nq = num_heads * embed_dim;
  for (int i = threadIdx.x; i < nq; i += blockDim.x) {
    const int head_idx = i / embed_dim;
    const int token_head = token_idx * query_stride + head_idx * head_size;
    const int token_out_head = token_idx * query_out_stride + head_idx * head_size;
    const int rot_offset = i % embed_dim;
    apply_dequant_rotary_embedding<scalar_t, IS_NEOX>(query + token_head, query_out + token_out_head, cos_ptr,
                                              sin_ptr, rot_offset, embed_dim, query_scale);
  }

  const int nk = num_kv_heads * embed_dim;
  for (int i = threadIdx.x; i < nk; i += blockDim.x) {
    const int head_idx = i / embed_dim;
    const int token_head = token_idx * key_stride + head_idx * head_size;
    const int token_out_head = token_idx * key_out_stride + head_idx * head_size;
    const int rot_offset = i % embed_dim;
    apply_dequant_rotary_embedding<scalar_t, IS_NEOX>(key + token_head, key_out + token_out_head, cos_ptr,
                                              sin_ptr, rot_offset, embed_dim, key_scale);
  }
}

} // namespace aphrodite

void rotary_embedding(
  torch::Tensor& positions,         // [batch_size, seq_len] or [num_tokens]
  torch::Tensor& query,             // [batch_size, seq_len, num_heads * head_size] or [num_tokens, num_heads * head_size]
  torch::Tensor& key,               // [batch_size, seq_len, num_kv_heads * head_size] or [num_tokens, num_kv_heads * head_size]
  int head_size,
  torch::Tensor& cos_sin_cache,     // [max_position, rot_dim]
  bool is_neox) {
  int num_tokens = query.numel() / query.size(-1);
  int rot_dim = cos_sin_cache.size(1);
  int num_heads = query.size(-1) / head_size;
  int num_kv_heads = key.size(-1) / head_size;
  int query_stride = query.stride(-2);
  int key_stride = key.stride(-2);
  dim3 grid(num_tokens);
  dim3 block(std::min(num_heads * rot_dim / 2, 512));
  const hipStream_t stream = at::cuda::getCurrentCUDAStream();
  APHRODITE_DISPATCH_FLOATING_TYPES(
    query.scalar_type(),
    "rotary_embedding",
    [&] {
      if (is_neox) {
        aphrodite::rotary_embedding_kernel<scalar_t, true><<<grid, block, 0, stream>>>(
          positions.data_ptr<int64_t>(),
          query.data_ptr<scalar_t>(),
          key.data_ptr<scalar_t>(),
          cos_sin_cache.data_ptr<scalar_t>(),
          rot_dim,
          query_stride,
          key_stride,
          num_heads,
          num_kv_heads,
          head_size);
      } else {
        aphrodite::rotary_embedding_kernel<scalar_t, false><<<grid, block, 0, stream>>>(
          positions.data_ptr<int64_t>(),
          query.data_ptr<scalar_t>(),
          key.data_ptr<scalar_t>(),
          cos_sin_cache.data_ptr<scalar_t>(),
          rot_dim,
          query_stride,
          key_stride,
          num_heads,
          num_kv_heads,
          head_size);
      }
    });
}


void invoke_dequant_rotary_embedding(
  torch::Tensor& positions,         // [num_tokens]
  torch::Tensor& query,             // [num_tokens, num_heads * head_size]
  torch::Tensor& query_out,             // [num_tokens, num_heads * head_size]
  torch::Tensor& key,               // [num_tokens, num_kv_heads * head_size]
  torch::Tensor& key_out,               // [num_tokens, num_kv_heads * head_size]
  int head_size,
  torch::Tensor& cos_sin_cache,     // [max_position, rot_dim]
  const float query_scale,
  const float key_scale,
  bool is_neox) {
  int num_tokens = query.size(0);
  int rot_dim = cos_sin_cache.size(1);
  int num_heads = query.size(1) / head_size;
  int num_kv_heads = key.size(1) / head_size;
  int query_stride = query.stride(0);
  int key_stride = key.stride(0);
  int query_out_stride = query_out.stride(0);
  int key_out_stride = key_out.stride(0);
  dim3 grid(num_tokens);
  dim3 block(std::min(num_heads * rot_dim / 2, 512));
  const hipStream_t stream = at::cuda::getCurrentCUDAStream();
  APHRODITE_DISPATCH_FLOATING_TYPES(
    query_out.scalar_type(),
    "dequant_rotary_embedding_kernel",
    [&] {
      if (is_neox) {
        aphrodite::dequant_rotary_embedding_kernel<scalar_t, true><<<grid, block, 0, stream>>>(
          positions.data_ptr<int64_t>(),
          query.data_ptr<int32_t>(),
          query_out.data_ptr<scalar_t>(),
          key.data_ptr<int32_t>(),
          key_out.data_ptr<scalar_t>(),
          cos_sin_cache.data_ptr<scalar_t>(),
          rot_dim,
          query_stride,
          query_out_stride,
          key_stride,
          key_out_stride,
          num_heads,
          num_kv_heads,
          head_size,
          query_scale,
          key_scale);
      } else {
        aphrodite::dequant_rotary_embedding_kernel<scalar_t, false><<<grid, block, 0, stream>>>(
          positions.data_ptr<int64_t>(),
          query.data_ptr<int32_t>(),
          query_out.data_ptr<scalar_t>(),
          key.data_ptr<int32_t>(),
          key_out.data_ptr<scalar_t>(),
          cos_sin_cache.data_ptr<scalar_t>(),
          rot_dim,
          query_stride,
          query_out_stride,
          key_stride,
          key_out_stride,
          num_heads,
          num_kv_heads,
          head_size,
          query_scale,
          key_scale);
      }
    });
}